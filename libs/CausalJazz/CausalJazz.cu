#include "CausalJazz.cuh"
#include "CudaEuler.cuh"
#include <iostream>

#define BLOCK_SIZE 128

#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

CausalJazz::CausalJazz()
: block_size(BLOCK_SIZE) {

}

CausalJazz::~CausalJazz() {

}

unsigned int CausalJazz::addDistribution(std::vector<double> _base, std::vector<double> _dims, std::vector<unsigned int> _res, std::vector<double> A) {
	grids.push_back(new CudaGrid(_base, _dims, _res, A));
	return grids.size() - 1;
}

void CausalJazz::buildJointDistributionFromChain(CudaGrid* A, CudaGrid* BgivenA, unsigned int out) {
	unsigned int numBlocks = (grids[out]->getTotalNumCells() + block_size - 1) / block_size;

	GenerateJointDistribution << <numBlocks, block_size >> > (
		grids[out]->getTotalNumCells(),
		grids[out]->getProbabilityMass(),
		A->getTotalNumCells(),
		A->getProbabilityMass(),
		BgivenA->getProbabilityMass());
}

void CausalJazz::buildJointDistributionFromFork(CudaGrid* A, CudaGrid* BgivenA, CudaGrid* CgivenA, unsigned int out) {
	unsigned int numBlocks = (grids[out]->getTotalNumCells() + block_size - 1) / block_size;

	GenerateJointDistributionFromFork << <numBlocks, block_size >> > (
		grids[out]->getTotalNumCells(),
		grids[out]->getProbabilityMass(),
		A->getTotalNumCells(),
		A->getProbabilityMass(),
		BgivenA->getRes()[1],
		BgivenA->getProbabilityMass(),
		CgivenA->getRes()[1],
		CgivenA->getProbabilityMass());
}

void CausalJazz::buildJointDistributionFromCollider(CudaGrid* A, CudaGrid* B, CudaGrid* CgivenAB, unsigned int out) {
	unsigned int numBlocks = (grids[out]->getTotalNumCells() + block_size - 1) / block_size;

	GenerateJointDistributionFromCollider << <numBlocks, block_size >> > (
		grids[out]->getTotalNumCells(),
		grids[out]->getProbabilityMass(),
		A->getTotalNumCells(),
		A->getProbabilityMass(),
		B->getTotalNumCells(),
		B->getProbabilityMass(),
		CgivenAB->getProbabilityMass());
}

void CausalJazz::buildMarginalDistribution(CudaGrid* A, unsigned int droppedDim, unsigned int out) {

	if (A->getNumDimensions() == 3) {
		unsigned int numBlocks = (grids[out]->getTotalNumCells() + block_size - 1) / block_size;

		if (droppedDim == 0) { // Drop A
			GenerateMarginalBC << <numBlocks, block_size >> > (
				grids[out]->getTotalNumCells(),
				grids[out]->getProbabilityMass(),
				A->getRes()[0],
				A->getRes()[1],
				A->getRes()[2],
				A->getProbabilityMass());
		}
		else if (droppedDim == 1) { // Drop B
			GenerateMarginalAC << <numBlocks, block_size >> > (
				grids[out]->getTotalNumCells(),
				grids[out]->getProbabilityMass(),
				A->getRes()[0],
				A->getRes()[1],
				A->getRes()[2],
				A->getProbabilityMass());
		}
		else if (droppedDim == 2) { // Drop C
			GenerateMarginalAB << <numBlocks, block_size >> > (
				grids[out]->getTotalNumCells(),
				grids[out]->getProbabilityMass(),
				A->getRes()[0],
				A->getRes()[1],
				A->getRes()[2],
				A->getProbabilityMass());
		}
	}
	else if (A->getNumDimensions() == 2) {
		unsigned int numBlocks = (grids[out]->getTotalNumCells() + block_size - 1) / block_size;

		if (droppedDim == 0) { // Drop A
			GenerateMarginalB << <numBlocks, block_size >> > (
				grids[out]->getTotalNumCells(),
				grids[out]->getProbabilityMass(),
				A->getRes()[0],
				A->getRes()[1],
				A->getProbabilityMass());
		}
		else if (droppedDim == 1) { // Drop B
			GenerateMarginalA << <numBlocks, block_size >> > (
				grids[out]->getTotalNumCells(),
				grids[out]->getProbabilityMass(),
				A->getRes()[0],
				A->getRes()[1],
				A->getProbabilityMass());
		}
	}
}

void CausalJazz::reduceJointDistributionToConditional(CudaGrid* A, unsigned int given, unsigned int out) {

}